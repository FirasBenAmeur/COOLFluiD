#include "hip/hip_runtime.h"
#include "FiniteVolumeCUDA/FVMCC_ComputeRhsJacobCell.hh"
#include "Framework/MeshData.hh"
#include "Framework/BlockAccumulatorBaseCUDA.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"
#include "FiniteVolume/CellData.hh"

#include "FiniteVolumeCUDA/FiniteVolumeCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"

#include "FiniteVolume/LaxFriedFlux.hh"
#include "FiniteVolume/LeastSquareP1PolyRec2D.hh"
#include "FiniteVolume/LeastSquareP1PolyRec3D.hh"
#include "FiniteVolume/BarthJesp.hh"
#include "FiniteVolume/Venktn2D.hh"

#include "MHD/MHD2DProjectionConsT.hh"
#include "MHD/MHD3DProjectionConsT.hh"
#include "MHD/MHD2DProjectionPrimT.hh"
#include "MHD/MHD3DProjectionPrimT.hh"
#include "MHD/MHDProjectionPrimToConsT.hh"
#include "MHD/MHD2DProjectionVarSet.hh"
#include "MHD/MHD3DProjectionVarSet.hh"
#include "FiniteVolumeMHD/LaxFriedFluxTanaka.hh"

#include "MultiFluidMHD/MultiFluidMHDVarSet.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsToRhoiViTiT.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMPlusUpFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMFluxMultiFluid.hh"

#include "Maxwell/Maxwell2DProjectionVarSet.hh"
#include "Maxwell/Maxwell2DProjectionConsT.hh"
#include "FiniteVolumeMaxwell/StegerWarmingMaxwellProjection2D.hh"

//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::MHD;
using namespace COOLFluiD::Physics::Maxwell;
using namespace COOLFluiD::Physics::MultiFluidMHD;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

  namespace Numerics {

    namespace FiniteVolume {

//////////////////////////////////////////////////////////////////////////////

#define FVMCC_MHD_RHS_JACOB_PROV(__dim__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeRhsJacobCell<LaxFriedFlux, \
						VarSetListT<MHD##__dim__##__svars__##T, MHD##__dim__##__uvars__##T>, \
						LeastSquareP1PolyRec##__dim__ , BarthJesp, __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsJacobMHD##__dim__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
FVMCC_MHD_RHS_JACOB_PROV(2D, ProjectionCons, ProjectionCons, 48, "CellNumJacobLaxFriedMHD2DCons")
FVMCC_MHD_RHS_JACOB_PROV(3D, ProjectionCons, ProjectionCons, 48, "CellNumJacobLaxFriedMHD3DCons")
FVMCC_MHD_RHS_JACOB_PROV(2D, ProjectionCons, ProjectionPrim, 48, "CellNumJacobLaxFriedMHD2DPrim")
FVMCC_MHD_RHS_JACOB_PROV(3D, ProjectionCons, ProjectionPrim, 48, "CellNumJacobLaxFriedMHD3DPrim")
#undef FVMCC_MHD_RHS_JACOB_PROV

#define FVMCC_MHD_RHS_JACOB_PROV_TANAKA(__dim__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeRhsJacobCell<LaxFriedFluxTanaka<MHD##__dim__##ProjectionVarSet>, \
						VarSetListT<MHD##__dim__##__svars__##T, MHD##__dim__##__uvars__##T>, \
						LeastSquareP1PolyRec##__dim__ , BarthJesp, __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsJacobMHDTanaka##__dim__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
FVMCC_MHD_RHS_JACOB_PROV_TANAKA(2D, ProjectionCons, ProjectionCons, 48, "CellNumJacobLaxFriedTanakaMHD2DCons")
FVMCC_MHD_RHS_JACOB_PROV_TANAKA(3D, ProjectionCons, ProjectionCons, 48, "CellNumJacobLaxFriedTanakaMHD3DCons")
FVMCC_MHD_RHS_JACOB_PROV_TANAKA(2D, ProjectionCons, ProjectionPrim, 48, "CellNumJacobLaxFriedTanakaMHD2DPrim")
FVMCC_MHD_RHS_JACOB_PROV_TANAKA(3D, ProjectionCons, ProjectionPrim, 48, "CellNumJacobLaxFriedTanakaMHD3DPrim")
#undef FVMCC_MHD_RHS_JACOB_PROV_TANAKA




//Provider for StegerWarmingProjectionMaxwell2D
#define FVMCC_MAXWELL_RHS_JACOB_PROV_STEGER(__dim__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeRhsJacobCell<StegerWarmingMaxwellProjection2D<Maxwell##__dim__##ProjectionVarSet>, \
						VarSetListT<Maxwell##__dim__##__svars__##T, Maxwell##__dim__##__uvars__##T>, \
						LeastSquareP1PolyRec##__dim__ , BarthJesp, __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsJacobMaxwellSteger##__dim__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
FVMCC_MAXWELL_RHS_JACOB_PROV_STEGER(2D, ProjectionCons, ProjectionCons, 48, "CellNumJacobStegerWarmingMaxwell2DCons")

#undef FVMCC_MAXWELL_RHS_JACOB_PROV_STEGER


// Provider for AUSMPlusUpFlux 
#define FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP(__dim__,__half__,__svars__,__uvars__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeRhsJacobCell<AUSMPlusUpFluxMultiFluid<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
			              VarSetListT<EulerMFMHD##__dim__##__half__##__svars__##T, EulerMFMHD##__dim__##__half__##__uvars__##T>, \
				      LeastSquareP1PolyRec##__dim__ , BarthJesp, __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsJacobMultiFluidMHDAUSMPlusUp##__dim__##__half__##__svars__##__uvars__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP(2D, Half,Cons, RhoiViTi, 48, "CellNumJacobAUSMPlusUpEulerMFMHD2DHalfRhoiViTi")

#undef FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP



//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr, 
				 CFreal* node, CFreal* nodePtr)
{
  // copy the state node data to shared memory
  for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////
      
template <typename PHYS>
HOST_DEVICE inline void setFaceNormal(FluxData<PHYS>* fd, CFreal* normal)
{  
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(normal);
  const CFreal area = n.norm2();
  fd->setFaceArea(area);
  const CFreal ovArea = 1./area;
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> un(fd->getUnitNormal());
  for (CFuint i = 0; i < PHYS::DIM; ++i) {
    un[i] = n[i]*ovArea;
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename PTR>
HOST_DEVICE void setFluxData(const CFuint f, const CFint stype, 
			     const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS>* fd,
			     PTR cellFaces)
{  
  fd->setStateID(RIGHT, stateID);
  CFreal* statePtrR = (stype > 0) ? &kd->states[stateID*PHYS::NBEQS] : &kd->ghostStates[stateID*PHYS::NBEQS];  
  CFreal* nodePtrR = (stype > 0) ? &kd->centerNodes[stateID*PHYS::DIM] : &kd->ghostNodes[stateID*PHYS::DIM];  
  setState<PHYS>(fd->getState(RIGHT), statePtrR, fd->getNode(RIGHT), nodePtrR);
  
  fd->setIsBFace(stype < 0);
  fd->setStateID(LEFT, cellID);
  const CFuint faceID = cellFaces[f*kd->nbCells + cellID];
  fd->setIsOutward(kd->isOutward[faceID] == cellID);
  
  CFreal* statePtrL = &kd->states[cellID*PHYS::NBEQS];
  CFreal* nodePtrL = &kd->centerNodes[cellID*PHYS::DIM];
  setState<PHYS>(fd->getState(LEFT), statePtrL, fd->getNode(LEFT), nodePtrL);
  setFaceNormal<PHYS>(fd, &kd->normals[faceID*PHYS::DIM]);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  std::cout << name << " = \t";
  for (CFuint i = 0; i < SIZE; ++i) {
    std::cout.precision(10); std::cout << array[i] << " ";
  }
  std::cout << "\n";
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void printArray(T* array) 
{
  for (CFuint i = 0; i < SIZE; ++i) {
    std::cout << array[i] << " ";
  }
  std::cout << "\n";
}

//////////////////////////////////////////////////////////////////////////////

template <typename MODEL>
HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
				     const CFreal* nodes, CFreal* midFaceCoord)
{  
  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
  coord = 0.;
  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
  for (CFuint n = 0; n < nbFaceNodes; ++n) {
    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
    const CFuint nodeID = cell->getNodeID(faceIdx,n);
    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
    for (CFuint d = 0; d < MODEL::DIM; ++d) {
      coord[d] += faceNode[d];
    }
  }
  coord *= ovNbFaceNodes;
}

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC>
__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				       const CFuint nbCells,
				       CFreal* states, 
				       CFreal* nodes,
				       CFreal* centerNodes,
				       CFreal* ghostStates,
				       CFreal* ghostNodes,
				       CFreal* uX,
				       CFreal* uY,
				       CFreal* uZ,
				       CFreal* limiter,
				       CFreal* updateCoeff, 
				       CFreal* rhs,
				       CFreal* normals,
				       CFint* isOutward,
				       const CFuint* cellInfo,
				       const CFuint* cellStencil,
				       const CFuint* cellFaces,
				       const CFuint* cellNodes,
				       const CFint*  neighborTypes,
				       const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) {    
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC, typename LIMITER>
__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				     const CFuint nbCells,
				     CFreal* states, 
				     CFreal* nodes,
				     CFreal* centerNodes,
				     CFreal* ghostStates,
				     CFreal* ghostNodes,
				     CFreal* uX,
				     CFreal* uY,
				     CFreal* uZ,
				     CFreal* limiter,
				     CFreal* updateCoeff, 
				     CFreal* rhs,
				     CFreal* normals,
				     CFint* isOutward,
				     const CFuint* cellInfo,
				     const CFuint* cellStencil,
				     const CFuint* cellFaces,
				     const CFuint* cellNodes,
				     const CFint*  neighborTypes,
				     const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
 
  if (cellID < nbCells) {    
    // compute all cell quadrature points at once (size of this array is overestimated)
    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    // compute cell-based limiter at once
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    LIMITER limt(dcol);
    
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
}
  
//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename POLYREC, typename LIMITER>
__global__ void computeFluxJacobianKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
					  typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
					  typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
					  typename NumericalJacobian::DeviceConfigOptions<typename SCHEME::MODEL>* dcon,
					  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
					  const CFuint nbCells,
					  const CFuint startCellID,
					  CFreal* states, 
					  CFreal* nodes,
					  CFreal* centerNodes,
					  CFreal* ghostStates,
					  CFreal* ghostNodes,
					  CFreal* blockJacob,
					  CFuint* blockStart,
					  CFreal* uX,
					  CFreal* uY,
					  CFreal* uZ,
					  CFreal* limiter,
					  CFreal* updateCoeff, 
					  CFreal* rhs,
					  CFreal* normals,
					  CFint* isOutward,
					  const CFuint* cellInfo,
					  const CFuint* cellStencil,
					  const CFuint* cellFaces,
					  const CFuint* cellNodes,
					  const CFint* neighborTypes,
					  const Framework::CellConn* cellConn)
{  
  typedef typename SCHEME::MODEL PHYS;
  
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x + startCellID;
  
  if (cellID < nbCells) {
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    SCHEME  fluxScheme(dcof);
    LIMITER limt(dcol);
    NumericalJacobian::DeviceFunc<PHYS> numJacob(dcon);
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, 
			   ghostNodes, updateCoeff, rhs, normals, uX, uY, uZ, isOutward);
    
    // compute all cell quadrature points at once (array size can be overestimated in 3D)
    const CFuint MAX_NB_FACES = PHYS::DIM*2;
    CFreal midFaceCoord[PHYS::DIM*MAX_NB_FACES];
    CudaEnv::CFVec<CFreal,PHYS::NBEQS> fluxDiff;
    CudaEnv::CFVec<CFreal,PHYS::NBEQS> resBkp;
    FluxData<PHYS> currFd; currFd.initialize();
    typename SCHEME::MODEL pmodel(dcop);
    
    // reset the rhs and update coefficients to 0
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    const CFuint nbRows = nbFacesInCell + 1;
    const CFuint bStartCellID = blockStart[cellID];
    
    // this block accumulator represents a column block (nbFaces+1 x 1)
    BlockAccumulatorBaseCUDA acc(nbRows, 1, PHYS::NBEQS, &blockJacob[bStartCellID]);
    acc.reset();
    
    // compute the face flux and flux numerical jacobian within the same loop
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      
      if (stype != 0) { // skip all partition faces
	const CFuint stateID = cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, &currFd, cellFaces);
	
	// compute face quadrature points (face centroids)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(&currFd, faceCenters, uX, uY, uZ, limiter);

        fluxScheme.prepareComputation(&currFd, &pmodel);
	fluxScheme(&currFd, &pmodel);

        for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  CFreal value = currFd.getResidual()[iEq];
	  res[iEq]   -= value;  // update the residual 
	  resBkp[iEq] = value;  // backup the current face-based residual
	}


	// update the update coefficient
	updateCoeff[cellID] += currFd.getUpdateCoeff();
		
	// only contribution from internal faces is computed here  
	if (stype > 0) { 	  
	  currFd.setIsPerturb(true);
	  // flux jacobian computation
	  for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar) {
	    // here we perturb the current variable for the left cell state
	    numJacob.perturb(iVar, &currFd.getState(LEFT)[iVar]);
	    
	    // extrapolate solution on quadrature points on both sides of the face
	    const CFreal rstateBkpL = currFd.getRstate(LEFT)[iVar];
	    polyRec.extrapolateOnFace(iVar, &currFd, faceCenters, uX, uY, uZ, limiter);
	    fluxScheme(&currFd, &pmodel); // compute the convective flux across the face
	    
	    // compute the numerical jacobian of the flux
	    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> resPert(currFd.getResidual());
	    numJacob.computeDerivative(&resBkp, &resPert, &fluxDiff);
	    
	    // contribution to the row corresponding of the current cell
	    // this subblock gets all contributions from all face cells
	    acc.addValues(0, 0, iVar, &fluxDiff[0]);
	    
	    // contribution to row corresponding to the f+1 cell: 
	    // this is the flux jacobian contribution for the neighbor cells
	    // due to the currently perturbed cell state and is opposite in sign
	    // because the outward normal for neighbors is inward for the current cell
	    fluxDiff *= -1.0;
	    acc.addValues(f+1, 0, iVar, &fluxDiff[0]); 
	    
	    // restore perturbed states
	    currFd.getRstate(LEFT)[iVar] = rstateBkpL;
	    numJacob.restore(&currFd.getState(LEFT)[iVar]);
	  }
	  
	  currFd.setIsPerturb(false);
	}
      }
    }
    
    
  }
}

//////////////////////////////////////////////////////////////////////////////
  
template <typename SCHEME, typename POLYREC, typename LIMITER>
void computeFluxJacobianCPU(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
			    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
			    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
			    typename NumericalJacobian::DeviceConfigOptions<typename SCHEME::MODEL>* dcon,
			    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
			    const CFuint nbCells,
			    CFreal* states, 
			    CFreal* nodes,
			    CFreal* centerNodes,
			    CFreal* ghostStates,
			    CFreal* ghostNodes, 
			    CFreal* blockJacob,
			    CFuint* blockStart,
			    CFreal* uX,
			    CFreal* uY,
			    CFreal* uZ,
			    CFreal* limiter,
			    CFreal* updateCoeff, 
			    CFreal* rhs,
			    CFreal* normals,
			    CFint* isOutward,
			    const CFuint* cellInfo,
			    const CFuint* cellStencil,
			    const CFuint* cellFaces,
			    const CFuint* cellNodes,
			    const CFint* neighborTypes,
			    const Framework::CellConn* cellConn)
{ 
  using namespace std;
  
  typedef typename SCHEME::MODEL PHYS;
  
  CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
  timer.start();
  
  FluxData<PHYS> fd; fd.initialize();
  FluxData<PHYS>* currFd = &fd;
  cf_assert(currFd != CFNULL);
  SCHEME fluxScheme(dcof);
  POLYREC polyRec(dcor);
  LIMITER limt(dcol);
  
  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			rhs, normals, uX, uY, uZ, isOutward);
  
  const CFuint MAX_NB_FACES = PHYS::DIM*2;
  CFreal midFaceCoord[PHYS::DIM*MAX_NB_FACES];
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> fluxDiff;
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> resBkp;
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
  NumericalJacobian::DeviceFunc<PHYS> numJacob(dcon);
  PHYS pmodel(dcop);
  
  // compute the cell-based gradients
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
  
  // printGradients<PHYS::NBEQS>(uX, uY, uZ, nbCells);
  CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::computeFluxJacobianCPU() => computing gradients took " << timer.elapsed() << " s\n");
  timer.start();
  
  // compute the cell based limiter
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // compute all cell quadrature points at once (size of this array is overestimated)
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    const CFuint cellID = cell.getCellID();
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      // compute cell-based limiter
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
  
  // printLimiter<PHYS::NBEQS>(limiter, nbCells);
  
  CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::computeFluxJacobianCPU() => computing limiter took " << timer.elapsed() << " s\n");
  timer.start();
  
  // compute the fluxes and the jacobian
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // reset the rhs and update coefficients to 0
    const CFuint cellID = cell.getCellID();
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    const CFuint nbRows = nbFacesInCell + 1;
    const CFuint bStartCellID = blockStart[cellID];
        
    // this block accumulator represents a column block (nbFaces+1 x 1)
    BlockAccumulatorBaseCUDA acc(nbRows, 1, PHYS::NBEQS, &blockJacob[bStartCellID]);
    acc.reset();
    
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      if (stype != 0) { // skip all partition faces
	const CFuint stateID =  cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
	
	// compute face quadrature points (centroid)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
		
        fluxScheme.prepareComputation(currFd, &pmodel); 
	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
        

	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  const CFreal value = currFd->getResidual()[iEq];
	  res[iEq]   -= value;  // update the residual 
	  resBkp[iEq] = value;  // backup the current face-based residual
	}
	
	// update the update coefficient
	updateCoeff[cellID] += currFd->getUpdateCoeff();
	
	// only contribution from internal faces is computed here  
	if (stype > 0) { 
	  currFd->setIsPerturb(true);
	  
	  // flux jacobian computation
	  for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar) {
	    // here we perturb the current variable for the left cell state
	    numJacob.perturb(iVar, &currFd->getState(LEFT)[iVar]);
	    
	    // extrapolate solution on quadrature points on both sides of the face
	    const CFreal rstateBkpL = currFd->getRstate(LEFT)[iVar];
	    polyRec.extrapolateOnFace(iVar, currFd, faceCenters, uX, uY, uZ, limiter);
	    fluxScheme(currFd, &pmodel); // compute the convective flux across the face
	    
	    // compute the numerical jacobian of the flux
	    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> resPert(currFd->getResidual());
	    numJacob.computeDerivative(&resBkp, &resPert, &fluxDiff);
	    
	    
	    
	    ///
	    /*if (cellID == 0 && stateID == 1 && iVar == 0) {
	      cout << "\n left 0, right 1 \n";
	      cout << iVar << " => resBkp   = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resBkp[0]);
	      cout << iVar << " => resPert  = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resPert[0]);
	      cout << iVar << " => fluxDiff = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&fluxDiff[0]);
	    }
	    
	    if (stateID == 0 && cellID == 1 && iVar == 0) {
	      cout << "\n left 1, right 0 \n";
	      cout << iVar << " => resBkp   = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resBkp[0]);
	      cout << iVar << " => resPert  = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resPert[0]);
	      cout << iVar << " => fluxDiff = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&fluxDiff[0]);
	      }*/
	    ///
	    
	    
	    
	    // flux is computed with the outward normal, so the sign is correct here
	    // contribution to the row corresponding of the current cell
	    // this subblock gets all contributions from all face cells
	    acc.addValues(0, 0, iVar, &fluxDiff[0]);
	    
	    // contribution to row corresponding to the f+1 cell: 
	    // this is the flux jacobian contribution for the neighbor cells
	    // due to the currently perturbed cell state and is opposite in sign
	    // because the outward normal for neighbors is inward for the current cell
	    fluxDiff *= -1.0;
	    
	    
	    ///
	    /* if (cellID == 0 && stateID == 1 && iVar == 0) {
	      cout << "\n left 0, right 1 \n";
	      cout << iVar << " => resBkp   = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resBkp[0]);
	      cout << iVar << " => resPert  = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resPert[0]);
	      cout << iVar << " => fluxDiff = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&fluxDiff[0]);
	    }
	    
	    if (stateID == 0 && cellID == 1 && iVar == 0) {
	      cout << "\n left 1, right 0 \n";
	      cout << iVar << " => resBkp   = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resBkp[0]);
	      cout << iVar << " => resPert  = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&resPert[0]);
	      cout << iVar << " => fluxDiff = "; std::cout.precision(12);printArray<CFreal,PHYS::NBEQS>(&fluxDiff[0]);
	      }*/
	    ///
	    
	    
	    acc.addValues(f+1, 0, iVar, &fluxDiff[0]);   
	    
	    // restore perturbed states
	    currFd->getRstate(LEFT)[iVar] = rstateBkpL;
	    numJacob.restore(&currFd->getState(LEFT)[iVar]);
	  }
	  	  
	  currFd->setIsPerturb(false);
	}

      }
    }
    //if (abs(res[6]) <= 1e-3){res[6] = 0.0;} 
  } 
  
  CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::computeFluxJacobianCPU() took " << timer.elapsed() << " s\n");
}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, typename POLYREC, typename LIMITER, CFuint NB_BLOCK_THREADS>
void FVMCC_ComputeRhsJacobCell<SCHEME,PHYSICS,POLYREC,LIMITER,NB_BLOCK_THREADS>::execute()
{
  using namespace std;
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() START\n");
  
  initializeComputationRHS();
  
  const CFuint nbCells = this->socket_states.getDataHandle().size();
  cf_assert(nbCells > 0);
  DataHandle<CFreal> updateCoeff = this->socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = this->socket_rhs.getDataHandle();
  DataHandle<CFreal> normals = this->socket_normals.getDataHandle();
  DataHandle<CFint> isOutward = this->socket_isOutward.getDataHandle();  
  
  SafePtr<SCHEME> lf = this->getMethodData().getFluxSplitter().template d_castTo<SCHEME>();
  SafePtr<POLYREC> pr = this->getMethodData().getPolyReconstructor().template d_castTo<POLYREC>();
  SafePtr<LIMITER> lm = this->getMethodData().getLimiter().template d_castTo<LIMITER>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();
  
  typedef typename SCHEME::template  DeviceFunc<GPU, PHYSICS> FluxScheme;  
  typedef typename POLYREC::template DeviceFunc<PHYSICS> PolyRec;  
  typedef typename LIMITER::template DeviceFunc<PHYSICS> Limiter;  
  
  CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
  
  if (this->m_onGPU) {
    
    timer.start();
    // copy of data that change at every iteration
    this->socket_states.getDataHandle().getGlobalArray()->put(); 
    this->m_ghostStates.put();
    
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<POLYREC, NOTYPE, GPU> dcor(pr);
    ConfigOptionPtr<LIMITER, NOTYPE, GPU> dcol(lm);
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    
    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() <blocksPerGrid, nThreads> = <" 
	  <<  blocksPerGrid << "," << nThreads << ">\n");
    
    //dim3 blocks(this->m_nbBlocksPerGridX, this->m_nbBlocksPerGridY);
    
    timer.start();
    
    // compute the cell-based gradients
    computeGradientsKernel<PHYSICS, PolyRec> <<<blocksPerGrid,nThreads>>> 
      (dcor.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => computeGradientsKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    
    // compute the limiter in each cell
    computeLimiterKernel<PHYSICS, PolyRec, Limiter> <<<blocksPerGrid,nThreads>>> 
      (dcol.getPtr(), 
       dcor.getPtr(), 
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => computeLimiterKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    // compute the flux jacobian in each cell
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => Configuring method \n");
    ConfigOptionPtr<NumericalJacobian, PHYSICS, GPU> dcon
      (&this->getMethodData().getNumericalJacobian());
    CFuint startCellID = 0;
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => End of Configuring method \n");
    
    CFreal FluxTime = 0.0;
    CFreal UpdateSystemTime = 0.0;
    
    for (CFuint s = 0; s < m_nbCellsInKernel.size(); ++s) {
      CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => loop " << s << " of " << m_nbCellsInKernel.size() << "\n");
      computeFluxJacobianKernel<FluxScheme, PolyRec, Limiter> <<<m_nbKernelBlocks,nThreads>>> 
	(dcof.getPtr(),
	 dcor.getPtr(),
	 dcol.getPtr(),
	 dcon.getPtr(),
	 dcop.getPtr(),
	 nbCells,
	 startCellID,
	 this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
	 this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
	 this->m_centerNodes.ptrDev(), 
	 this->m_ghostStates.ptrDev(),
	 this->m_ghostNodes.ptrDev(),
	 m_blockJacobians.ptrDev(), 
	 m_blockStart.ptrDev(),
	 this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
	 updateCoeff.getLocalArray()->ptrDev(), 
	 rhs.getLocalArray()->ptrDev(),
	 normals.getLocalArray()->ptrDev(),
	 isOutward.getLocalArray()->ptrDev(),
	 this->m_cellInfo.ptrDev(),
	 this->m_cellStencil.ptrDev(),
	 this->m_cellFaces->getPtr()->ptrDev(),
	 this->m_cellNodes->getPtr()->ptrDev(),
	 this->m_neighborTypes.ptrDev(),
	 this->m_cellConn.ptrDev());

      FluxTime += timer.elapsed();
      timer.start();
      CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => m_blockJacobians.get() \n");
      m_blockJacobians.get();
      // update the portion of system matrix computed by this kernel
      CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => updateSystemMatrix(" << s <<") \n");
      updateSystemMatrix(s);
      startCellID += m_nbCellsInKernel[s];
      UpdateSystemTime += timer.elapsed();
      timer.start();
    }
    
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => computeFluxJacobianKernel took " << FluxTime << " s\n");
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => updateSystemMatrix took took " << UpdateSystemTime << " s\n");
    
    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();

    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
  }
  else {
    ConfigOptionPtr<SCHEME>  dcof(lf);
    ConfigOptionPtr<POLYREC> dcor(pr);
    ConfigOptionPtr<LIMITER> dcol(lm);
    ConfigOptionPtr<NumericalJacobian, PHYSICS> dcon(&this->getMethodData().getNumericalJacobian());
    ConfigOptionPtr<typename PHYSICS::PTERM> dcop(phys);

    computeFluxJacobianCPU<FluxScheme, PolyRec, Limiter>
      (dcof.getPtr(),
       dcor.getPtr(),
       dcol.getPtr(),
       dcon.getPtr(),
       dcop.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptr(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptr(),
       this->m_centerNodes.ptr(), 
       this->m_ghostStates.ptr(),
       this->m_ghostNodes.ptr(),
       m_blockJacobians.ptr(), 
       m_blockStart.ptr(),
       this->socket_uX.getDataHandle().getLocalArray()->ptr(),
       this->socket_uY.getDataHandle().getLocalArray()->ptr(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptr(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptr(),
       updateCoeff.getLocalArray()->ptr(), 
       rhs.getLocalArray()->ptr(),
       normals.getLocalArray()->ptr(),
       isOutward.getLocalArray()->ptr(),
       this->m_cellInfo.ptr(),
       this->m_cellStencil.ptr(),
       this->m_cellFaces->getPtr()->ptr(),
       this->m_cellNodes->getPtr()->ptr(),
       this->m_neighborTypes.ptr(),
       this->m_cellConn.ptr());
    
    // update the system matrix
    timer.start();
    updateSystemMatrix(0);
    CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => updateSystemMatrix took " << timer.elapsed() << " s\n");
  }
  
  timer.start();
  // compute flux jacobians on boundaries
  executeBC();
  CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() => executeBC() took " << timer.elapsed() << " s\n");

  finalizeComputationRHS();
  
  CFLog(VERBOSE, "FVMCC_ComputeRhsJacobCell::execute() END\n");
 
 
/*
     const CFuint nbEqs = PhysicalModelStack::getActive()->getNbEq();
     //DataHandle<CFreal> rhs = this->socket_rhs.getDataHandle();
     DataHandle<State*, GLOBAL> states = this->socket_states.getDataHandle();
     //DataHandle<CFreal> updateCoeff = this->socket_updateCoeff.getDataHandle();
     CFreal* state = this->socket_states.getDataHandle().getGlobalArray()->ptr();
   for (CFuint iState = 0; iState < states.size(); ++iState) {
   // if (iState == 2299){
     CFLog(VERBOSE, " \t iState: " << iState << "\t UpdateCoeff: " << updateCoeff[iState] << "\n");
     for (CFuint iEq = 0; iEq < nbEqs; ++iEq) {
       //cout.precision(14); cout.setf(ios::scientific,ios::floatfield); cout << rhs(iState, iEq, nbEqs);
       CFLog(VERBOSE, rhs(iState, iEq, nbEqs) << " \n \t" << state[iState*nbEqs + iEq] << " \n");
       //CFLog(VERBOSE, rhs(iState, iEq, nbEqs) << "\n");
     }
     cout << endl;
   // }
   }
*/



 /* for (int i = 0; i < updateCoeff.size(); ++i) {
    // for (int i = 0; i < 10000; ++i) {
    std::cout << "updateCoeff[" << i << "] = " << updateCoeff[i]  << std::endl;
    std::cout << "rhs[" << i << "] = ";
    for (int j = 0; j < 9; ++j) {
      std::cout << rhs[i*9+j] << " ";
    }
    std::cout << std::endl;
  }*/
  
  // abort();
  
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

template <CFuint NBEQS>
void printGradients(CFreal* uX, CFreal* uY, CFreal* uZ, CFuint nbCells)
{  
  CFuint idxr = 0;
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    for (CFuint i = 0; i < NBEQS; ++i, ++idxr) {
      std::cout << "cellID["<< cellID << "], "<< i << " => UX (";
      std::cout.precision(12); std::cout << uX[idxr] << ", " << uY[idxr] << ", " << uZ[idxr] << ")\n";
    }
  } 
}

//////////////////////////////////////////////////////////////////////////////

template <CFuint NBEQS>
void printLimiter(CFreal* limiter, CFuint nbCells)
{ 
  CFuint idxl = 0;
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    std::cout << "cellID["<< cellID << "] => LIM (";
    for (CFuint i = 0; i < NBEQS; ++i, ++idxl) {
      std::cout.precision(12); std::cout << limiter[idxl] << " ";
    }
    std::cout << ")\n";
  }
}
 
//////////////////////////////////////////////////////////////////////////////

   } // namespace FiniteVolume
    
  } // namespace Numerics

} // namespace COOLFluiD
